#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>


const int ARRAY_SIZE = 64;

__global__ void square(float * d_out, float * d_in) {
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f*f;

}


void gpuCode() {
	
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}

	float h_out[ARRAY_SIZE];

	float * d_in;	// input array on GPU
	float * d_out;	// output array on GPU

	//allocate memory on GPU
	hipMalloc((void **)&d_in, ARRAY_BYTES);
	hipMalloc((void**)&d_out, ARRAY_BYTES);

	//copy input data from CPU to GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	//execute calculation on GPU
	square << <1, ARRAY_SIZE >> >  (d_out, d_in); 
												  
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
}

int main(int argc,char** argv)
{
	gpuCode();
    return 0;
}

